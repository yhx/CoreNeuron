#include "hip/hip_runtime.h"

#include "fireinfo"

__device__ int commitFireInfo(FireInfo *shared_buf, volatile unsigned int size, FireInfo *global_buf, int *global_size, int offset) 
{
	__shared__ volatile unsigned int start_loc;
	if (threadIdx.x == 0) {
		start_loc = atomicAdd(global_size, (int)size);
	}
	__syncthreads();

	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		global_buf[offset + start_loc + idx].n = shared_buf[idx].n;
		global_buf[offset + start_loc + idx].time = shared_buf[idx].time;
	}

	return 0;
}

