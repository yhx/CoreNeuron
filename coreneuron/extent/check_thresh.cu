#include "hip/hip_runtime.h"


#include "extent.h"

__device__ int commit2globalTable(int *shared_buf, volatile unsigned int size, int *global_buf, int * global_size, int offset) 
{
	__shared__ volatile unsigned int start_loc;
	if (threadIdx.x == 0) {
		start_loc = atomicAdd(global_size, (int)size);
	}
	__syncthreads();

	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		global_buf[offset + start_loc + idx] = shared_buf[idx];
	}

	return 0;
}


__global__  record_netcon(NetCon *n, PreSyn *pre, int *nids, int size) {

	double time_a[MAX_BLOCK_SIZE];
	NetCon 
	int block_num = gridDim.x;
	int num_per_block = (idx_size-1+block_num)/block_num;
	int num_per_block_1 = num_per_block - 1;
	int offset = size - block_num * num_per_block_1;

	int idx_size_t = 0;
	int idx_offset = 0;
	int block_idx = blockIdx.x;

	if (block_idx < offset) {
		idx_size_t = num_per_block;
		idx_offset = block_idx * num_per_block;
	} else if (block_idx < block_num) {
		idx_size_t = num_per_block_1;
		idx_offset = offset * num_per_block + (block_idx-offset) * num_per_block_1;
	} else {
		idx_size_t = 0;
	}

	int tid = threadIdx.x;
	int thread_num = blockDim.x;

	for  (int i=0; i < idx_size_t; i++) {
		PreSyn *p = pre + nids[i];
		for (int j=tid; j<p->nc_cnt_; j+=thread_num) {
			NetCon *d = netcon_in_presyn_order[p->nc_index_+j];
			if (d->active_ && d->target_) {


			}
		}
	}
}
