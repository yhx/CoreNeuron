#include "hip/hip_runtime.h"


#include "connection.h"

__device__ int commit2globalTable(int *shared_buf, volatile unsigned int size, int *global_buf, int * global_size, int offset) 
{
	__shared__ volatile unsigned int start_loc;
	if (threadIdx.x == 0) {
		start_loc = atomicAdd(global_size, (int)size);
	}
	__syncthreads();

	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		global_buf[offset + start_loc + idx] = shared_buf[idx];
	}

	return 0;
}

__global__  gen_spike(FireInfo *spike, int *s_size, PreSyn *pre, int *nids, int size, double time) {

	__shared__ FireInfo spike_a[MAX_BLOCK_SIZE];
	__shared__ volatile int count;

	if (threadIdx.x == 0) {
		count == 0;
	}
	__syncthreads();

	int block_num = gridDim.x;
	int num_per_block = (idx_size-1+block_num)/block_num;
	int num_per_block_1 = num_per_block - 1;
	int offset = size - block_num * num_per_block_1;

	int idx_size_t = 0;
	int idx_offset = 0;
	int block_idx = blockIdx.x;

	if (block_idx < offset) {
		idx_size_t = num_per_block;
		idx_offset = block_idx * num_per_block;
	} else if (block_idx < block_num) {
		idx_size_t = num_per_block_1;
		idx_offset = offset * num_per_block + (block_idx-offset) * num_per_block_1;
	} else {
		idx_size_t = 0;
	}

	int tid = threadIdx.x;
	int thread_num = blockDim.x;

	int count_t = 0;
	bool commit = false;

	for  (int i=0; i < idx_size_t; i++) {
		PreSyn *p = pre + nids[i];
		for (int j=tid; j<p->nc_cnt_; j+=thread_num) {
			NetCon *d = netcon_in_presyn_order[p->nc_index_+j];
			commit = d->active_ && d->target_;
			if (commit) {
				count_t = atomicAdd((int *)&count, 1);
				if (count_t < MAX_BLOCK_SIZE) {
					spike_a[count_t].time = time + d->delay_;
					spike_a[count_t].n =  d;
					commit = false;
				} else {
					commitFireInfo(spike_a, MAX_BLOCK_SIZE, spike, &s_size[0], 0);
				}
			}
			__syncthreads();

			if (threadIdx.x == 0 && count >= MAX_BLOCK_SIZE) {
				count == 0;
			}
			__syncthreads();
			if (commit) {
				count_t = atomicAdd((int *)&count, 1);
				if (count_t < MAX_BLOCK_SIZE) {
					spike_a[count_t].time = time + d->delay_;
					spike_a[count_t].n =  d;
					commit = false;
				}
			}
			__syncthreads();
			if (count > 0) {
				commitFireInfo(spike_a, MAX_BLOCK_SIZE, spike, &s_size[0], 0);
			}
			if (threadIdx.x == 0) {
				count == 0;
			}
			__syncthreads();
		}
	}
}

__global__  fire_spike(FireInfo *spike, int *s_size, FireInfo *spike1, int s1_size, FireInfo *spike2, int *s2_size, double time) {
	__shared__ FireInfo spike_a[MAX_BLOCK_SIZE];
	__shared__ FireInfo spike_b[MAX_BLOCK_SIZE];
	__shared__ volatile int count_a;
	__shared__ volatile int count_b;

	if (threadIdx.x == 0) {
		count_a == 0;
		count_b == 0;
	}

	bool commit_a = false, commit_b = false;
	int count_a_t = 0, count_b_t = 0;
	__syncthreads();
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	for (int id=tid; id<s_size[0]; id+=blockDim.x*gridDim.x) {
		commit_a = spike[id].time <= time;
		commit_b = ~commit_a;
		if (commit_a) {
			count_a_t = atomicAdd((int *)&count_a, 1);
			if (count_a_t < MAX_BLOCK_SIZE) {
				spike_a[count_a_t].time = time + d->delay_;
				spike_a[count_a_t].n =  d;
				commit_a = false;
			} else {
				commitFireInfo(spike_a, MAX_BLOCK_SIZE, spike1, &s1_size[0], 0);
			}
		}
		__syncthreads();

		if (threadIdx.x == 0 && count >= MAX_BLOCK_SIZE) {
			count_a == 0;
		}
		__syncthreads();
		if (commit) {
			count_a_t = atomicAdd((int *)&count, 1);
			if (count_a_t < MAX_BLOCK_SIZE) {
				spike_a[count_a_t].time = time + d->delay_;
				spike_a[count_a_t].n =  d;
				commit = false;
			}
		}
		__syncthreads();
		if (count_a > 0) {
			commitFireInfo(spike_a, MAX_BLOCK_SIZE, spike1, &s1_size[0], 0);
		}
		if (threadIdx.x == 0) {
			count_a == 0;
		}
		__syncthreads();

		if (commit_b) {
			count_b_t = atomicAdd((int *)&count_a, 1);
			if (count_b_t < MAX_BLOCK_SIZE) {
				spike_b[count_a_t].time = time + d->delay_;
				spike_b[count_a_t].n =  d;
				commit_b = false;
			} else {
				commitFireInfo(spike_b, MAX_BLOCK_SIZE, spike2, &s2_size[0], 0);
			}
		}
		__syncthreads();

		if (threadIdx.x == 0 && count >= MAX_BLOCK_SIZE) {
			count_b == 0;
		}
		__syncthreads();
		if (commit) {
			count_b_t = atomicAdd((int *)&count_b, 1);
			if (count_b_t < MAX_BLOCK_SIZE) {
				spike_b[count_a_t].time = time + d->delay_;
				spike_b[count_a_t].n =  d;
				commit = false;
			}
		}
		__syncthreads();
		if (count_b > 0) {
			commitFireInfo(spike_b, MAX_BLOCK_SIZE, spike2, &s2_size[0], 0);
		}
		if (threadIdx.x == 0) {
			count_b == 0;
		}
		__syncthreads();

	}
}


__global__  reset_size(int *s_size, FireInfo *spike1, int s1_size, FireInfo *spike2, int *s2_size, double time) {
